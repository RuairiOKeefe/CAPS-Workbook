#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES

//The maximum particles to be simulated
#define MAX_PARTICLES 256
//How many simulations are to be ran
#define NUM_SIMULATIONS 1000
//How many tests are to be ran
#define NUM_TESTS 100
//The delta time between each simulation
#define TIMESTEP 0.01f;
//Small, near 0 value to improve result
#define SOFTENING 1e-4f
//Newtons gravitational constant, probably wont use this because of how weak gravity is 
#define G 6.673e-11f

//Density of hydrogen in kg/m3
#define H_DENISTY 0.08988
//Density of oxygen in kg/m3
#define O_DENISTY 1.429
//Density of iron in kg/m3
#define FE_DENISTY 7874.0
//Density of osmium in kg/m3
#define OS_DENISTY 22600.0

#include "GLShader.h"
#include <GLFW\glfw3.h>
#include <chrono>
#include <glm\gtc\type_ptr.hpp>
#include <glm\gtc\matrix_transform.hpp>
#include <glm\common.hpp>
#include <glm\gtx\norm.hpp>
#include "Camera.h"
#include <fstream>
#include <random>
#include <cmath>
#include <chrono>
#include <iostream>
#include <ctime>
#include <math.h>
#include <algorithm>
#include "Texture.h"
#include <omp.h>
#include <thread>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//The texture for each particle
Texture tex;
//Basic target camera 
Camera cam;

//Particle shader
GLShader shader;
GLFWwindow* window;

// Uniform locations for shader.
GLuint CameraRight_worldspace_ID;
GLuint CameraUp_worldspace_ID;
GLuint ViewProjMatrixID;

// Get the number of threads this hardware can support.
int numThreads = std::thread::hardware_concurrency();

// This class represents the particle.
struct Particle
{
	//Position of the particle.
	glm::vec3 pos;
	//Colour of the particle.
	unsigned char r, g, b, a;
	//Radius of the particle in meters
	float radius;
	//Velocity of the particle
	glm::vec3 velocity;
	//Particles mass in kg
	float mass;
};

GLuint VertexArrayID;
static GLfloat* gl_pos_data = new GLfloat[MAX_PARTICLES * 4];
static GLubyte* gl_colour_data = new GLubyte[MAX_PARTICLES * 4];
GLuint pos_buffer;
GLuint colour_buffer;
GLuint vertex_buffer;

//Array of all the particles in the scene
Particle particles[MAX_PARTICLES];
//The positions of every particle after each simulation
Particle particleMovements[NUM_SIMULATIONS][MAX_PARTICLES];
double lastTime;

unsigned long long particlesSize;
Particle *particlesBuffer;

using namespace std::chrono;

void LoadParticles()
{
	for (int i = 0; i < MAX_PARTICLES; i++)
	{
		double x = (rand() % 100) - 50;
		double y = (rand() % 100) - 50;
		double z = (rand() % 100) - 50;

		particles[i].pos = glm::dvec3(x, y, z);
		particles[i].velocity = glm::dvec3(0);
		particles[i].r = 0;
		particles[i].g = 100;
		particles[i].b = 255;
		particles[i].a = 255;
		particles[i].mass = 1;

		//if (i == 0)
		//particles[i].mass = 100;

		//Volume = mass/density
		float volume = particles[i].mass / H_DENISTY;
		particles[i].radius = cbrt((3 * volume) / (4 * M_PI));

		gl_colour_data[4 * i + 0] = particles[i].r;
		gl_colour_data[4 * i + 1] = particles[i].g;
		gl_colour_data[4 * i + 2] = particles[i].b;
		gl_colour_data[4 * i + 3] = particles[i].a;

	}

	hipMemcpy(particlesBuffer, &particles, particlesSize, hipMemcpyHostToDevice);

	for (int i = 0; i < MAX_PARTICLES; i++)
	{
		particleMovements[0][i] = particles[i];
	}
}

int Initialise()
{
	// Initialise GLFW
	if (!glfwInit())
	{
		fprintf(stderr, "Failed to initialize GLFW\n");
		getchar();
		return -1;
	}

	glfwWindowHint(GLFW_SAMPLES, 4);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	// Open a window and create its OpenGL context
	window = glfwCreateWindow(1920, 1080, "N-Body Simulation", NULL, NULL);
	if (window == NULL) {
		fprintf(stderr, "Failed to open GLFW window.\n");
		getchar();
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);

	// Initialize GLEW
	glewExperimental = true; // Needed for core profile
	if (glewInit() != GLEW_OK) {
		fprintf(stderr, "Failed to initialize GLEW\n");
		getchar();
		glfwTerminate();
		return -1;
	}

	// Ensure we can capture the escape key being pressed below
	glfwSetInputMode(window, GLFW_STICKY_KEYS, GL_TRUE);
	// Hide the mouse and enable unlimited mouvement
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

	// Set the mouse at the center of the screen
	glfwPollEvents();
	glfwSetCursorPos(window, 1024 / 2, 768 / 2);

	//Backgroud colour
	glClearColor(0.2f, 0.2f, 0.2f, 0.0f);

	glGenVertexArrays(1, &VertexArrayID);
	glBindVertexArray(VertexArrayID);


	// Create and compile our GLSL program from the shaders
	shader.SetProgram();
	shader.AddShaderFromFile("../res/shaders/Quad.vert", GLShader::VERTEX);
	shader.AddShaderFromFile("../res/shaders/Quad.frag", GLShader::FRAGMENT);
	shader.Link();

	cam.SetProjection(glm::quarter_pi<float>(), 1920 / 1080, 2.414f, 100000);
	cam.SetWindow(window);
	cam.SetPosition(glm::vec3(0, 0, 200));

	// Vertex shader
	CameraRight_worldspace_ID = glGetUniformLocation(shader.GetId(), "CameraRight_worldspace");
	CameraUp_worldspace_ID = glGetUniformLocation(shader.GetId(), "CameraUp_worldspace");
	ViewProjMatrixID = glGetUniformLocation(shader.GetId(), "VP");

	lastTime = glfwGetTime();

	tex = Texture("../res/textures/Particle.png");

	static const GLfloat g_vertex_buffer_data[] =
	{
		-0.5f, -0.5f, 0.0f,
		0.5f, -0.5f, 0.0f,
		-0.5f,  0.5f, 0.0f,
		0.5f,  0.5f, 0.0f,
	};
	glGenBuffers(1, &vertex_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, vertex_buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(g_vertex_buffer_data), g_vertex_buffer_data, GL_STATIC_DRAW);

	//The VBO containing the positions and sizes of the particles
	glGenBuffers(1, &pos_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, pos_buffer);
	//Initialize with empty (NULL) buffer : it will be updated later, each frame.
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 4 * sizeof(GLfloat), NULL, GL_STREAM_DRAW);

	//The VBO containing the colors of the particles
	glGenBuffers(1, &colour_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, colour_buffer);
	//Initialize with empty (NULL) buffer : it will be updated later, each frame.
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 4 * sizeof(GLubyte), NULL, GL_STREAM_DRAW);

	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	hipSetDevice(0);
	particlesSize = sizeof(Particle)*MAX_PARTICLES;
	hipMalloc((void**)&particlesBuffer, particlesSize);

	LoadParticles();

	return 0;
}

__global__
void CalculateForces(Particle* particles)
{
	float fX = 0.0f; float fY = 0.0f; float fZ = 0.0f;

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	for (int j = 0; j < MAX_PARTICLES; j++)
	{
		float dx = particles[j].pos.x - particles[i].pos.x;
		float dy = particles[j].pos.y - particles[i].pos.y;
		float dz = particles[j].pos.z - particles[i].pos.z;
		float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
		float invDist = 1.0f / sqrtf(distSqr);
		float invDist3 = invDist * invDist * invDist;

		fX += (particles[i].mass * particles[j].mass) * dx * invDist3;
		fY += (particles[i].mass * particles[j].mass) * dy * invDist3;
		fZ += (particles[i].mass * particles[j].mass) * dz * invDist3;
	}

	particles[i].velocity.x += fX;
	particles[i].velocity.y += fY;
	particles[i].velocity.z += fZ;
}

void SimulateParticles(int currentIndex)
{
	//move somewhere more logical
	int numBlocks = 16;

	hipMemcpy(particlesBuffer, &particles, particlesSize, hipMemcpyHostToDevice);
	CalculateForces << <numBlocks, MAX_PARTICLES / numBlocks >> > (particlesBuffer);
	hipDeviceSynchronize();
	hipMemcpy(particles, &particlesBuffer[0], particlesSize, hipMemcpyDeviceToHost);

	for (int i = 0; i < MAX_PARTICLES; i++)
	{
		Particle& p = particles[i];
		p.pos += p.velocity * TIMESTEP;
		particleMovements[currentIndex][i] = particles[i];
	}
}

void UpdatePosBuffer(int currentIndex)
{
	Particle tempParticles[MAX_PARTICLES];
	for (int i = 0; i < MAX_PARTICLES; i++)
	{
		tempParticles[i] = particleMovements[currentIndex][i];
	}

	bool swap = 1;
	for (int i = 1; (i <= MAX_PARTICLES) && swap; i++)
	{
		swap = 0;
		for (int j = 0; j < (MAX_PARTICLES - 1); j++)
		{
			Particle& p1 = tempParticles[j];
			Particle& p2 = tempParticles[j + 1];
			if (glm::distance(p2.pos, cam.GetPosition()) > glm::distance(p1.pos, cam.GetPosition()))
			{
				Particle temp = p1;
				p1 = p2;
				p2 = temp;
				swap = 1;
			}
		}
	}

	for (int i = 0; i < MAX_PARTICLES; i++)
	{
		Particle& p = tempParticles[i];

		// Update GPU buffer with new positions.
		gl_pos_data[4 * i + 0] = p.pos.x;
		gl_pos_data[4 * i + 1] = p.pos.y;
		gl_pos_data[4 * i + 2] = p.pos.z;
		gl_pos_data[4 * i + 3] = p.radius;

	}
}

void Update(double deltaTime)
{
	//make targe once I set bounds
	float ratio_width = glm::quarter_pi<float>() / static_cast<float>(1920);
	float ratio_height = glm::quarter_pi<float>() / static_cast<float>(1080);

	double xpos, ypos;
	glfwGetCursorPos(window, &xpos, &ypos);
	glfwSetCursorPos(window, 1920.0 / 2, 1080.0 / 2);
	// Calculate delta of cursor positions from last frame
	double delta_x = xpos - 1920.0 / 2;
	double delta_y = ypos - 1080.0 / 2;
	// Multiply deltas by ratios - gets actual change in orientation
	delta_x *= ratio_width;
	delta_y *= ratio_height;
	cam.Rotate(static_cast<float>(delta_x), static_cast<float>(-delta_y)); // flipped y to revert the invert.
	cam.Update(deltaTime);

}


void Render()
{
	// Update the OpenGL buffers with updated particle positions.
	glBindBuffer(GL_ARRAY_BUFFER, pos_buffer);
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 4 * sizeof(GLfloat), NULL, GL_STREAM_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, MAX_PARTICLES * sizeof(GLfloat) * 4, gl_pos_data);

	glBindBuffer(GL_ARRAY_BUFFER, colour_buffer);
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 4 * sizeof(GLubyte), NULL, GL_STREAM_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, MAX_PARTICLES * sizeof(GLubyte) * 4, gl_colour_data);

	// Clear the screen
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	//	glClearColor(1, 1, 1, 1);
	glm::mat4 ProjectionMatrix = cam.GetProjection();
	glm::mat4 ViewMatrix = cam.GetView();
	glm::mat4 ViewProjectionMatrix = ProjectionMatrix * ViewMatrix;

	// Use our shader
	shader.Use();
	glUniform3f(CameraRight_worldspace_ID, ViewMatrix[0][0], ViewMatrix[1][0], ViewMatrix[2][0]);
	glUniform3f(CameraUp_worldspace_ID, ViewMatrix[0][1], ViewMatrix[1][1], ViewMatrix[2][1]);
	glUniformMatrix4fv(ViewProjMatrixID, 1, GL_FALSE, &ViewProjectionMatrix[0][0]);


	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, tex.id);
	glUniform1i(glGetUniformLocation(shader.GetId(), "tex"), 1);


	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, vertex_buffer);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);

	// 2nd attribute buffer : positions of particles' centers
	glEnableVertexAttribArray(1);
	glBindBuffer(GL_ARRAY_BUFFER, pos_buffer);
	glVertexAttribPointer(1, 4, GL_FLOAT, GL_FALSE, 0, (void*)0);

	// 3rd attribute buffer : particles' colors
	glEnableVertexAttribArray(2);
	glBindBuffer(GL_ARRAY_BUFFER, colour_buffer);
	glVertexAttribPointer(2, 4, GL_UNSIGNED_BYTE, GL_TRUE, 0, (void*)0);


	glVertexAttribDivisor(0, 0);
	glVertexAttribDivisor(1, 1);
	glVertexAttribDivisor(2, 1);
	glDrawArraysInstanced(GL_TRIANGLE_STRIP, 0, 4, MAX_PARTICLES);

	glDisableVertexAttribArray(0);
	glDisableVertexAttribArray(1);
	glDisableVertexAttribArray(2);

	// Swap buffers
	glfwSwapBuffers(window);
	glfwPollEvents();
}


int main(void)
{
	if (Initialise() == -1)
		return -1;
	std::ofstream data((std::to_string(MAX_PARTICLES) + "P_" + std::to_string(NUM_SIMULATIONS) + "S_" + std::to_string(NUM_TESTS) + "T.csv").c_str(), std::ofstream::out);
	for (int n = 0; n < NUM_TESTS; n++)
	{
		clock_t t;
		t = clock();
		for (int i = 0; i < NUM_SIMULATIONS; i++)
		{
			SimulateParticles(i);
		}
		clock_t end = clock();
		float elapsedTime = float(end - t) / CLOCKS_PER_SEC;
		data << elapsedTime << std::endl;
		LoadParticles();
	}
	data.close();

	int i = 0;
	//While still running and esc hasnt been pressed
	while (glfwGetKey(window, GLFW_KEY_ESCAPE) != GLFW_PRESS && glfwWindowShouldClose(window) == 0)
	{
		UpdatePosBuffer(i);
		double currentTime = glfwGetTime();
		double delta = currentTime - lastTime;
		Update(delta);
		Render();
		lastTime = currentTime;
		i++;
		if (i > NUM_SIMULATIONS)
			i = 0;
	}

	delete[] gl_pos_data;

	//Cleanup VBO and shader
	glDeleteBuffers(1, &colour_buffer);
	glDeleteBuffers(1, &pos_buffer);
	glDeleteBuffers(1, &vertex_buffer);
	glDeleteProgram(shader.GetId());
	glDeleteVertexArrays(1, &VertexArrayID);


	//Close OpenGL window and terminate GLFW
	glfwTerminate();

	return 0;
}